#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include<stdbool.h>
#include<hip/hip_runtime.h>


// Result from last compute of world.
unsigned char *g_resultData=NULL;

// Current state of world. 
unsigned char *g_data=NULL;

// Current width of world.
size_t g_worldWidth=0;

/// Current height of world.
size_t g_worldHeight=0;

/// Current data length (product of width and height)
size_t g_dataLength=0;  // g_worldWidth * g_worldHeight

static inline void gol_initAllZeros( size_t worldWidth, size_t worldHeight )
{   
    int i;

    g_worldWidth = worldWidth;
    g_worldHeight = worldHeight;
    g_dataLength = g_worldWidth * g_worldHeight;


    hipMallocManaged( &g_data, (g_dataLength * sizeof(unsigned char)));
    hipMallocManaged( &g_resultData, (g_dataLength * sizeof(unsigned char))); 
    // set all rows of world to zero
    for( i = 0; i < g_dataLength; i++)
    {
        g_data[i] = 0;
        g_resultData[i] = 0;
    }

}

static inline void gol_initAllOnes( size_t worldWidth, size_t worldHeight )
{
    size_t i;
    g_worldWidth = worldWidth;
    g_worldHeight = worldHeight;
    g_dataLength = g_worldWidth * g_worldHeight;

    hipMallocManaged( &g_data, (g_dataLength * sizeof(unsigned char)));
    hipMallocManaged( &g_resultData, (g_dataLength * sizeof(unsigned char))); 
    // set all rows of world to true
    for( i = 0; i < g_dataLength; i++)
    {
        g_data[i] = 1;
        g_resultData[i] = 1;
    }
    
}

static inline void gol_initOnesInMiddle( size_t worldWidth, size_t worldHeight )
{
    size_t i;
    
    g_worldWidth = worldWidth;
    g_worldHeight = worldHeight;
    g_dataLength = g_worldWidth * g_worldHeight;

    hipMallocManaged( &g_data, (g_dataLength * sizeof(unsigned char)));
    hipMallocManaged( &g_resultData, (g_dataLength * sizeof(unsigned char))); 

    // set first 1 rows of world to true
    for( i = 10*g_worldWidth; i < 11*g_worldWidth; i++)
    {
	    if( (i >= ( 10*g_worldWidth + 10)) && (i < (10*g_worldWidth + 20)))
	    {
            g_data[i] = 1;
            g_resultData[i] = 1;
        }
        else
        {
            g_data[i] = 0;
            g_resultData[i] = 0;
        }

    }
    
}

static inline void gol_initOnesAtCorners( size_t worldWidth, size_t worldHeight )
{   
    size_t i;
    g_worldWidth = worldWidth;
    g_worldHeight = worldHeight;
    g_dataLength = g_worldWidth * g_worldHeight;
    hipMallocManaged( &g_data, (g_dataLength * sizeof(unsigned char)));
    hipMallocManaged( &g_resultData, (g_dataLength * sizeof(unsigned char))); 

    // set all rows of world to zero
    for( i = 0; i < g_dataLength; i++)
    {
        g_data[i] = 0;
        g_resultData[i] = 0;
    }
    g_data[0] = 1; // upper left
    g_data[worldWidth-1]=1; // upper right
    g_data[(worldHeight * (worldWidth-1))]=1; // lower left
    g_data[(worldHeight * (worldWidth-1)) + worldWidth-1]=1; // lower right
    
    g_resultData[0] = 1; // upper left
    g_resultData[worldWidth-1]=1; // upper right
    g_resultData[(worldHeight * (worldWidth-1))]=1; // lower left
    g_resultData[(worldHeight * (worldWidth-1)) + worldWidth-1]=1; // lower right

}

static inline void gol_initSpinnerAtCorner( size_t worldWidth, size_t worldHeight )
{   
    size_t i;
    g_worldWidth = worldWidth;
    g_worldHeight = worldHeight;
    g_dataLength = g_worldWidth * g_worldHeight;

    hipMallocManaged( &g_data, (g_dataLength * sizeof(unsigned char)));
    hipMallocManaged( &g_resultData, (g_dataLength * sizeof(unsigned char))); 
    // set all rows of world to zero
    for( i = 0; i < g_dataLength; i++)
    {
        g_data[i] = 0;
        g_resultData[i] = 0;
    }
    g_data[0] = 1; // upper left
    g_data[1] = 1; // upper left +1
    g_data[worldWidth-1]=1; // upper right
    g_resultData[0] = 1; // upper left
    g_resultData[1] = 1; // upper left +1
    g_resultData[worldWidth-1]=1; // upper right
}

static inline void gol_initMaster( unsigned int pattern, size_t worldWidth, size_t worldHeight )
{
    switch(pattern)
    {
    case 0:
	gol_initAllZeros( worldWidth, worldHeight );
	break;
	
    case 1:
	gol_initAllOnes( worldWidth, worldHeight );
	break;
	
    case 2:
	gol_initOnesInMiddle( worldWidth, worldHeight );
	break;
	
    case 3:
	gol_initOnesAtCorners( worldWidth, worldHeight );
	break;

    case 4:
	gol_initSpinnerAtCorner( worldWidth, worldHeight );
	break;

    default:
	printf("Pattern %u has not been implemented \n", pattern);
	exit(-1);
    }
}

static inline void gol_swap( unsigned char **pA, unsigned char **pB)
{
    // Swap the pointers of pA and pB.
    unsigned char *temp = *pA;
    *pA = *pB;
    *pB = temp;

}
 
__device__ unsigned int gol_countAliveCells(unsigned char* data, 
					   size_t x0, 
					   size_t x1, 
					   size_t x2, 
					   size_t y0, 
					   size_t y1,
					   size_t y2) 
{
  
    // You write this function - it should return the number of alive cell for data[x1+y1]
    // There are 8 neighbors - see the assignment description for more details.
    int aliveCell = 0;
    // up left
    aliveCell += data[x0 + y0];
    // up 
    aliveCell += data[x1 + y0];
    // up right
    aliveCell += data[x2 + y0];
    // left
    aliveCell += data[x0 + y1];
    // buttom left
    aliveCell += data[x0 + y2];
    // buttom
    aliveCell += data[x1 + y2];
    // buttom right
    aliveCell += data[x2 + y2];
    // right
    aliveCell += data[x2 + y1];
    return aliveCell;
}


// Don't modify this function or your submitty autograding may incorrectly grade otherwise correct solutions.
static inline void gol_printWorld()
{
    int i, j;

    for( i = 0; i < g_worldHeight; i++)
    {
	printf("Row %2d: ", i);
	for( j = 0; j < g_worldWidth; j++)
	{
	    printf("%u ", (unsigned int)g_data[(i*g_worldWidth) + j]);
	}
	printf("\n");
    }

    printf("\n\n");
}

// This function is added to compute the state of current cell given alive neighbors number
__device__ unsigned int gol_computeState(unsigned int aliveCell, unsigned char currentState)
{
    switch(currentState)
    {
    case 0:
        // Any dead cell with exactly three live neighbors becomes a live cell, as if by reproduction
        if (aliveCell == 3)
            return 1;
        else
            return 0;
        break;
    case 1:
        //Any live cell with fewer than two live neighbors dies, as if caused by under-population.
        if (aliveCell < 2)
            return 0;
        //Any live cell with two or three live neighbors lives on to the next generation.
        if (aliveCell <= 3)
            return 1;
        //Any live cell with more than three live neighbors dies, as if by over-population.
        else
            return 0;
        break;
    
    default:
        printf("Cell state %u is not 0 or 1 \n", currentState);
        return 0;
    }
}

// CUDA kernel function to compute the whole world for one step
__global__ void gol_kernel( unsigned char* d_data,
                            unsigned int worldWidth,
                            unsigned int worldHeight,
                            unsigned char* d_resultData)
{
    size_t x, y, i;
    size_t y0, y1, y2;
    size_t x0, x2;
    unsigned int aliveCells;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int worldSize = worldHeight * worldWidth;
    for(i = index; i < worldSize; i+=stride)
    {   
        x = i % worldWidth;
        y = i / worldWidth;
        x0 = (x + worldWidth - 1) % worldWidth;
        x2 = (x + 1) % worldWidth;
        y0 = ((y + worldHeight - 1) % worldHeight) * worldWidth;
        y1 = y * worldWidth;
        y2 = ((y + 1) % worldHeight) * worldWidth;
        aliveCells = gol_countAliveCells(d_data, x0, x, x2, y0, y1, y2);
        d_resultData[y1 + x] = gol_computeState(aliveCells, d_data[y1 + x]);
    }

}
// This function computes the world via a CUDA kernel
bool gol_kernelLaunch(  unsigned char** d_data,
                        unsigned char** d_resultData,
                        size_t worldWidth,
                        size_t worldHeight,
                        size_t iterationsCount,
                        ushort threadsCount)
{
    int i;
    for(i = 0; i<iterationsCount; ++i)
    {
        gol_kernel<<<1, threadsCount>>>(*d_data, worldWidth, worldHeight, *d_resultData);
        gol_swap(d_data, d_resultData);
    }
    
    hipDeviceSynchronize();
    return 1;
}



int main(int argc, char *argv[])
{
    unsigned int pattern = 0;
    unsigned int worldSize = 0;
    unsigned int itterations = 0;
    unsigned int threadPerBlock = 0;
    unsigned int outputOn = 0;

    printf("This is the Game of Life running in parallel on a GPU.\n");

    if( argc != 6 )
    {
    printf("CUDA GOL requires 5 arguments: pattern number, sq size of the world and the number of itterations, "
    "threads per block and output-on/off e.g. ./gol 4 64 2 2 0\n");
	exit(-1);
    }

    pattern = atoi(argv[1]);
    worldSize = atoi(argv[2]);
    itterations = atoi(argv[3]);
    threadPerBlock = atoi(argv[4]);
    outputOn = atoi(argv[5]);

    
    gol_initMaster(pattern, worldSize, worldSize);
    
    // gol_iterateSerial( itterations );
    gol_kernelLaunch(&g_data, &g_resultData, worldSize, worldSize, itterations, threadPerBlock);

    if (outputOn>0)
    {
    printf("######################### FINAL WORLD IS ###############################\n");
    gol_printWorld();
    }
    hipFree(g_data);
    hipFree(g_resultData);
    
    return 0;
}